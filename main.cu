#include <iostream>
#include <fstream>
#include <stdlib.h>
#include <unistd.h>
#include <vector>
#include <complex>
#include <sys/types.h>
#include <sys/stat.h>
#include <string.h>
#include <math.h>
#include <map>
#include <stdexcept>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

//#include <helper_functions.h>
//#include <helper_cuda.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"CUDA_ERROR:\ncode:%s\nfile: %s\nline:%d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

///////////////////////////////////////////////////////////////////////////////////////////////
////
////    Constants & Typedefs
////
///////////////////////////////////////////////////////////////////////////////////////////////

#define PI 3.14159265359
#define BLOCK_SIZE 4096
#define SIGNAL_THRESHOLD 200
#define MAX_TRANSMISSIONS 200

//172MHz gives us CB
#define SAMPLE_RATE 172089331.259
#define BATCH_SIZE 1
#define NUM_STREAMS 2
#define BLOCKS_PER_STREAM 16

#define HzInMHz 1000000

typedef char byte;
typedef float2 Complex;

///////////////////////////////////////////////////////////////////////////////////////////////
////
////    Device Variables
////
///////////////////////////////////////////////////////////////////////////////////////////////

__device__ int transmissionCount;

__device__ int timeStep;

///////////////////////////////////////////////////////////////////////////////////////////////
////
////    Kernels
////
///////////////////////////////////////////////////////////////////////////////////////////////

void __device__ createTransmission( int idx ,
		int* transmissionBins,
		hipfftReal* scaledResultBuffer,
		hipfftReal* transmissionFrequencies,
		hipfftReal* transmissionStarts,
		hipfftReal* transmissionStrengths,
		bool* activeTransmissions
	)
{

	transmissionBins[ transmissionCount - 1 ] = idx;

	//frequency in MHz
	transmissionFrequencies[ transmissionCount - 1 ] = idx * SAMPLE_RATE / BLOCK_SIZE / HzInMHz;

	transmissionStarts[ transmissionCount - 1 ] = timeStep / SAMPLE_RATE;

	transmissionStrengths[ transmissionCount - 1 ] = scaledResultBuffer[ idx ];

	activeTransmissions[ idx ] = true;

}

void __device__ finishTransmission( int idx,
		int* transmissionBins,
		hipfftReal* transmissionEnds,
		bool* activeTransmissions
	)
{

	for( int i = transmissionCount - 1 ; i >= 0 ; i-- )
	{

		if( transmissionBins[ i ] == idx )
		{

			transmissionEnds[ i ] = timeStep / SAMPLE_RATE;

			activeTransmissions[ idx ] = false;

			return;

		}

	}

}

void __global__ scaleResult( hipfftReal* scaledResultBuffer , hipfftComplex* resultBuffer )
{

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if( idx < BLOCK_SIZE )
	{

		scaledResultBuffer[ idx ] = sqrt( resultBuffer[ idx ].x * resultBuffer[ idx ].x * +
										  resultBuffer[ idx ].y * resultBuffer[ idx ].y      );

		scaledResultBuffer[ idx ] = 20 * log10( scaledResultBuffer[ idx ] );

	}

}

void __global__ initTransmissionArray( bool* activeTransmissions )
{

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	transmissionCount = 0;

	if( idx < BLOCK_SIZE )

		activeTransmissions[ idx ] = false;

}

void __global__ findTransmissions(
		hipfftReal* scaledResultBuffer ,
		int* deviceBins,
		hipfftReal *deviceFrequencies,
		hipfftReal *deviceStarts,
		hipfftReal *deviceEnds,
		hipfftReal *deviceStrengths,
		bool* activeTransmissions
	)
{

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if( idx < BLOCK_SIZE && idx != 0 )
	{

		if( scaledResultBuffer[ idx ] > SIGNAL_THRESHOLD && activeTransmissions[ idx ] == false )
		{

			atomicAdd( &transmissionCount , 1 );

			createTransmission( idx , deviceBins , scaledResultBuffer , deviceFrequencies , deviceStarts , deviceStrengths, activeTransmissions );

		}

		if( scaledResultBuffer[ idx ] < SIGNAL_THRESHOLD && activeTransmissions[ idx ] == true )
		{

			finishTransmission( idx , deviceBins , deviceEnds , activeTransmissions );

		}

	}

	//timeStep += BLOCK_SIZE;

	atomicAdd( &timeStep , 1 );

}

///////////////////////////////////////////////////////////////////////////////////////////////
////
////    Device Pointers
////
///////////////////////////////////////////////////////////////////////////////////////////////

hipfftComplex *deviceResult = 0;

hipfftReal *deviceSource = 0;

hipfftReal *deviceScaledResult = 0;

int* deviceBins = 0;

hipfftReal *deviceFrequencies = 0;

hipfftReal *deviceStarts = 0;

hipfftReal *deviceEnds = 0;

hipfftReal *deviceStrengths = 0;

bool* deviceActiveTransmissions = 0;

int* deviceCount = 0;

///////////////////////////////////////////////////////////////////////////////////////////////
////
////    Host Variables
////
///////////////////////////////////////////////////////////////////////////////////////////////

int* hostBins;

hipfftReal *hostFrequencies;

hipfftReal *hostStarts;

hipfftReal *hostEnds;

hipfftReal *hostStrengths;

///////////////////////////////////////////////////////////////////////////////////////////////
////
////    Functions
////
///////////////////////////////////////////////////////////////////////////////////////////////

int main( int argc , char** argv )
{
	
	std::string filename = std::string( argv[1] );

	std::ifstream f;

	struct stat filestatus;

	stat( filename.c_str() , &filestatus );

	size_t filesize = filestatus.st_size;

	f.open( filename.c_str() , std::ios::in | std::ios::binary );

	if( !f.good() )
	{
	
		std::cerr << "Can't open file" << std::endl;

		exit( 1 );

	}

	hipfftReal* original = 0;

	//std::cout << "1" << std::endl;

	//cudaSetDevice( 1 );

	gpuErrchk( hipHostMalloc( (void**) &original , filesize * sizeof( hipfftReal ) , hipHostMallocDefault) );

	//std::cout << "2" << std::endl;

	for( unsigned int i = 0 ; i < filesize ; i++ )
	{
	
		original[i] = (hipfftReal) (byte) f.get();

	}

	f.close();

	hipGetSymbolAddress( (void**) &deviceCount , HIP_SYMBOL(transmissionCount) );

	gpuErrchk( hipMalloc( &deviceSource , NUM_STREAMS * BLOCKS_PER_STREAM * BLOCK_SIZE * sizeof(hipfftReal) ));

	gpuErrchk( hipMalloc( &deviceResult ,  NUM_STREAMS * BLOCKS_PER_STREAM * BLOCK_SIZE * sizeof(hipfftComplex) ));

	gpuErrchk( hipMalloc( &deviceScaledResult ,  NUM_STREAMS * BLOCKS_PER_STREAM * BLOCK_SIZE * sizeof(hipfftReal) ));

	gpuErrchk( hipMalloc( &deviceBins , MAX_TRANSMISSIONS * sizeof(int) ));

	gpuErrchk( hipMalloc( &deviceFrequencies ,  MAX_TRANSMISSIONS * sizeof(hipfftReal) ));

	gpuErrchk( hipMalloc( &deviceStarts , MAX_TRANSMISSIONS * sizeof(hipfftReal) ));

	gpuErrchk( hipMalloc( &deviceEnds ,  MAX_TRANSMISSIONS * sizeof(hipfftReal) ));

	gpuErrchk( hipMalloc( &deviceStrengths ,  MAX_TRANSMISSIONS * sizeof(hipfftReal) ));

	gpuErrchk( hipMalloc( &deviceActiveTransmissions ,  NUM_STREAMS * BLOCKS_PER_STREAM * BLOCK_SIZE * sizeof(bool) ));

	//std::cout << "3" << std::endl;

	initTransmissionArray<<< 64 , 32 >>>( deviceActiveTransmissions );

	//std::cout << "4" << std::endl;

	hipStream_t streams[ NUM_STREAMS ];

	for( int i = 0 ; i < NUM_STREAMS ; i++ )
	{

		gpuErrchk( hipStreamCreate( &streams[ i ] ) );

	}

	//std::cout << "5" << std::endl;

	//prepare the FFT
	hipfftHandle plans[ NUM_STREAMS ];

	hipfftResult_t fft_result;

	for( int i = 0 ; i < NUM_STREAMS ; i++ )
	{

		fft_result = hipfftPlan1d( &plans[i] , BLOCK_SIZE , HIPFFT_R2C , BATCH_SIZE );

		if( fft_result != HIPFFT_SUCCESS )

			exit(1);

		fft_result = hipfftSetStream( plans[i] , streams[i] );

		if( fft_result != HIPFFT_SUCCESS )

			exit(1);

	}

	//std::cout << "6" << std::endl;

	for( unsigned int j = 0 ; j < filesize * 0.25 - BLOCK_SIZE * NUM_STREAMS  ; j += BLOCK_SIZE * NUM_STREAMS * BLOCKS_PER_STREAM )
	{
		int iteration_offset = j * sizeof( hipfftReal );
		
		for( int k = 0 ; k < NUM_STREAMS ; k++ )
		{

			int stream_offset = k * BLOCK_SIZE * BLOCKS_PER_STREAM;

			hipMemcpyAsync( deviceSource + stream_offset  , original + iteration_offset + stream_offset , BLOCK_SIZE * BLOCKS_PER_STREAM , hipMemcpyHostToDevice , streams[ k ] );

			for( int l = 0 ; l < BLOCKS_PER_STREAM ; l++ )
			{

				int block_offset = l * BLOCK_SIZE;

				fft_result = hipfftExecR2C( plans[k] , deviceSource + stream_offset + block_offset , deviceResult + stream_offset + block_offset );

				if( fft_result != HIPFFT_SUCCESS )

					exit(2);

				// num blocks * num threads = fftsize / 2 ... nyquist limit
				scaleResult<<< 64 , 32 , 0 , streams[ k ] >>>( deviceScaledResult + stream_offset + block_offset , deviceResult + stream_offset + block_offset );

				gpuErrchk( hipPeekAtLastError() );

				findTransmissions<<< 64 , 32 , 0 , streams[ k ] >>>(
						deviceScaledResult + stream_offset + block_offset,
						deviceBins,
						deviceFrequencies,
						deviceStarts,
						deviceEnds,
						deviceStrengths,
						deviceActiveTransmissions
					);

				//std::cout << "11" << std::endl;

				gpuErrchk( hipPeekAtLastError() );

			}
		}

	}

	//std::cout << "12" << std::endl;

	//Copy all that crap back
	int* hostCount = new int;

	hostBins = new int[ MAX_TRANSMISSIONS ];

	hostFrequencies = new hipfftReal[ MAX_TRANSMISSIONS ];

	hostStarts = new hipfftReal[ MAX_TRANSMISSIONS ];

	hostEnds = new hipfftReal[ MAX_TRANSMISSIONS ];

	hostStrengths = new hipfftReal[ MAX_TRANSMISSIONS ];

	//std::cout << "13" << std::endl;

	gpuErrchk( hipMemcpy( hostBins , deviceBins , MAX_TRANSMISSIONS * sizeof( int ) , hipMemcpyDeviceToHost ));

	gpuErrchk( hipMemcpy( hostFrequencies , deviceFrequencies , MAX_TRANSMISSIONS * sizeof( hipfftReal ) , hipMemcpyDeviceToHost ));

	gpuErrchk( hipMemcpy( hostStarts , deviceStarts , MAX_TRANSMISSIONS * sizeof( hipfftReal ) , hipMemcpyDeviceToHost ));

	gpuErrchk( hipMemcpy( hostEnds , deviceEnds , MAX_TRANSMISSIONS * sizeof( hipfftReal ) , hipMemcpyDeviceToHost ));

	gpuErrchk( hipMemcpy( hostStrengths , deviceStrengths , MAX_TRANSMISSIONS * sizeof( hipfftReal ) , hipMemcpyDeviceToHost ));

	gpuErrchk( hipMemcpy( hostCount , deviceCount , sizeof( int ) , hipMemcpyDeviceToHost ) );

	//std::cout << "14" << std::endl;

	std::cout << *hostCount << std::endl;

	std::ofstream fo;

	fo.open( "spikes.txt" );

	for( unsigned int i = 0 ; i < *hostCount ; i++ )
	{

		fo << "==== TRANSMISSION ====" << "\n";

		//In MHz
		fo << "Bin             : " << hostBins[ i ] << " \n";
		fo << "Frequency       : " << hostFrequencies[ i ] << " MHz\n";
		fo << "Signal strength : " << hostStrengths[ i ] << " dB\n";
		fo << "Time start      : " << hostStarts[ i ] << " s\n";
		fo << "Time end        : " << hostEnds[ i ] << " s\n";

	}

	return 0;

}
