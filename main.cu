#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <stdlib.h>
#include <unistd.h>
#include <vector>
#include <complex>
#include <sys/types.h>
#include <sys/stat.h>
#include <string.h>
#include <math.h>
#include <map>
#include <stdexcept>
#include <hipfft/hipfft.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#define PI 3.14159265359
#define BLOCK_SIZE 4096
#define SIGNAL_THRESHOLD 102
#define MAX_TRANSMISSIONS 200

//172MHz gives us CB
#define SAMPLE_RATE 172089331.259
#define BATCH_SIZE 1

#define HzInMHz 1000000

typedef char byte;
typedef float2 Complex;

__device__ hipfftReal* sourceBuffer;

__device__ hipfftComplex* resultBuffer;

__device__ hipfftReal* scaledResultBuffer;

__device__ bool* activeTransmissions;

__device__ int* transmissionBins;

__device__ hipfftReal* transmissionFrequencies;

__device__ hipfftReal* transmissionStarts;

__device__ hipfftReal* transmissionEnds;

__device__ hipfftReal* transmissionStrengths;

__device__ int transmissionCount;

__device__ int timeStep;

void __global__ scaleResult( )
{

	int idx = threadIdx.x;

	if( idx < BLOCK_SIZE )
	{

		scaledResultBuffer[ idx ] = sqrt( resultBuffer[ idx ][ 0 ] * resultBuffer[ idx ][ 0 ] * +
										  resultBuffer[ idx ][ 1 ] * resultBuffer[ idx ][ 1 ]      );

		scaledResultBuffer[ idx ] = 20 * log10( scaledResultBuffer[ idx ] );

	}

}

void __global__ findTransmissions( )
{

	int idx = threadIdx.x;

	if( idx < BLOCK_SIZE )
	{

		if( scaledResultBuffer[ idx ] > SIGNAL_THRESHOLD && activeTransmissions[ idx ] == false )
		{



		}

	}

}

void __device__ createTransmission( int idx )
{



}

hipfftComplex *deviceResult;

hipfftReal *deviceSource;

hipfftReal *deviceScaledResult;

int* deviceBins;

hipfftReal *deviceFrequencies;

hipfftReal *deviceStarts;

hipfftReal *deviceEnds;

hipfftReal *deviceStrengths;

bool* deviceActiveTransmissions;

int* deviceCount;

void outputFFTData( std::string filename, fftw_real* data , unsigned int size );

class transmission
{

public:

	int bin;

	float frequency;

	float timeStart;

	float timeEnd;

	float peakStrength;

};

int main( int argc , char** argv )
{
	
	std::string filename = std::string( argv[1] );

	std::ifstream f;

	std::map< int , transmission > currentSpikes;

	std::vector< transmission > historicalSpikes;

	struct stat filestatus;

	stat( filename.c_str() , &filestatus );

	size_t filesize = filestatus.st_size;

	f.open( filename.c_str() , std::ios::in | std::ios::binary );

	if( !f.good() )
	{
	
		std::cerr << "Can't open file" << std::endl;

		exit( 1 );

	}

	hipfftReal* original = new hipfftReal[ filesize ];

	for( unsigned int i = 0 ; i < filesize ; i++ )
	{
	
		original[i] = (hipfftReal) (byte) f.get();

	}

	f.close();

	int fft_size = BLOCK_SIZE;

	int max_transmissions = MAX_TRANSMISSIONS;

	//get the address for the device's source buffer
	hipGetSymbolAddress( (void**) &deviceSource , sourceBuffer );

	//get the address for the device's result buffer
	hipGetSymbolAddress( (void**) &deviceResult , resultBuffer );

	//get the address for the device's result buffer
	hipGetSymbolAddress( (void**) &deviceScaledResult , scaledResultBuffer );

	//get the address for the device's result buffer
	hipGetSymbolAddress( (void**) &deviceBins , transmissionBins );

	//get the address for the device's result buffer
	hipGetSymbolAddress( (void**) &deviceFrequencies , transmissionFrequencies );

	//get the address for the device's result buffer
	hipGetSymbolAddress( (void**) &deviceStarts , transmissionStarts );

	//get the address for the device's result buffer
	hipGetSymbolAddress( (void**) &deviceEnds , transmissionEnds );

	//get the address for the device's result buffer
	hipGetSymbolAddress( (void**) &deviceStrengths , transmissionStrengths );

	//get the address for the device's result buffer
	hipGetSymbolAddress( (void**) &deviceCount , transmissionCount );

	//get the address for the device's result buffer
	hipGetSymbolAddress( (void**) &deviceActiveTransmissions , activeTransmissions );

	hipMalloc( &deviceSource , filesize * sizeof(hipfftReal) );

	hipMalloc( &deviceResult ,  fft_size * sizeof(hipfftComplex) );

	hipMalloc( &deviceScaledResult ,  fft_size * sizeof(hipfftReal) );

	hipMalloc( &deviceBins , max_transmissions * sizeof(int) );

	hipMalloc( &deviceFrequencies ,  max_transmissions * sizeof(hipfftReal) );

	hipMalloc( &deviceStarts , max_transmissions * sizeof(hipfftReal) );

	hipMalloc( &deviceEnds ,  max_transmissions * sizeof(hipfftReal) );

	hipMalloc( &deviceStrengths ,  max_transmissions * sizeof(hipfftReal) );

	hipMalloc( &deviceActiveTransmissions ,  fft_size * sizeof(bool) );

	// TODO: This giant memcpy will become a pipelined streaming thingy
	hipMemcpy( deviceSource , original , filesize * 0.25 * sizeof( hipfftReal ) , hipMemcpyHostToDevice );

	for( unsigned int j = 0 ; j < filesize * 0.25 - fft_size  ; j += fft_size )
	{
		
		//prepare the FFT
		hipfftHandle p;

		hipfftPlan1d( &p , BLOCK_SIZE , HIPFFT_R2C , BATCH_SIZE );

		//Run the FFT
		hipfftExecR2C( p , deviceSource, deviceResult );

		//calculate amplitude of first N/2 bins (Nyquist Limit?)
		for( unsigned int i = 0 ; i < fft_size / 2 ; i++ )
		{

			bool activeTransmission = true;

			try
			{

				currentSpikes.at( i );

			}
			catch( std::out_of_range& e )
			{

				activeTransmission = false;

			}

			if( resultScaled[ i ] > SIGNAL_THRESHOLD && activeTransmission == false )
			{
				
				transmission trans;

				trans.bin = i;

				//frequency in MHz
				trans.frequency = i * SAMPLE_RATE / fft_size / HzInMHz;

				trans.timeStart = j / SAMPLE_RATE;

				trans.peakStrength = resultScaled[ i ];

				currentSpikes.insert( std::pair< int, transmission >( i , trans ) );

				//debug
				outputFFTData( "spikeWindow.txt" , resultScaled , fft_size );

			}

			if( resultScaled[ i ] < SIGNAL_THRESHOLD && activeTransmission == true )
			{

				transmission t = currentSpikes.at( i );

				t.timeEnd = j / SAMPLE_RATE;

				historicalSpikes.push_back( t );

				currentSpikes.erase( i );

			}
		}

	}

	std::ofstream fo;

	fo.open( "spikes.txt" );

	for( unsigned int i = 0 ; i < historicalSpikes.size() ; i++ )
	{

		fo << "====TRANSMISSION====" << "\n";

		//In MHz
		fo << "Frequency       : " << historicalSpikes[ i ].frequency << " MHz\n";
		fo << "Signal strength : " << historicalSpikes[ i ].peakStrength << " dB\n";
		fo << "Time start      : " << historicalSpikes[ i ].timeStart << " s\n";
		fo << "Time end        : " << historicalSpikes[ i ].timeEnd << " s\n";

	}

	return 0;

}

void outputFFTData( std::string filename, fftw_real* data , unsigned int size )
{
	std::ofstream fo;

	fo.open( filename.c_str() );

	for( unsigned int i = 0 ; i < size * 0.5 ; i++ )
	{

		fo << data[ i ] << std::endl;

	}

}
