#include <iostream>
#include <fstream>
#include <stdlib.h>
#include <unistd.h>
#include <vector>
#include <complex>
#include <sys/types.h>
#include <sys/stat.h>
#include <string.h>
#include <math.h>
#include <map>
#include <stdexcept>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

//#include <helper_functions.h>
//#include <helper_cuda.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"CUDA_ERROR:\ncode:%s\nfile: %s\nline:%d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

///////////////////////////////////////////////////////////////////////////////////////////////
////
////    Constants & Typedefs
////
///////////////////////////////////////////////////////////////////////////////////////////////

#define PI 3.14159265359
#define BLOCK_SIZE 4096
#define SIGNAL_THRESHOLD 200
#define MAX_TRANSMISSIONS 200

//172MHz gives us CB
#define SAMPLE_RATE 172089331.259
#define BATCH_SIZE 1

#define HzInMHz 1000000

typedef char byte;
typedef float2 Complex;

///////////////////////////////////////////////////////////////////////////////////////////////
////
////    Device Variables
////
///////////////////////////////////////////////////////////////////////////////////////////////

__device__ int transmissionCount;

__device__ int timeStep;

///////////////////////////////////////////////////////////////////////////////////////////////
////
////    Kernels
////
///////////////////////////////////////////////////////////////////////////////////////////////

void __device__ createTransmission( int idx ,
		int* transmissionBins,
		hipfftReal* scaledResultBuffer,
		hipfftReal* transmissionFrequencies,
		hipfftReal* transmissionStarts,
		hipfftReal* transmissionStrengths,
		bool* activeTransmissions
	)
{

	transmissionBins[ transmissionCount - 1 ] = idx;

	//frequency in MHz
	transmissionFrequencies[ transmissionCount - 1 ] = idx * SAMPLE_RATE / BLOCK_SIZE / HzInMHz;

	transmissionStarts[ transmissionCount - 1 ] = timeStep / SAMPLE_RATE;

	transmissionStrengths[ transmissionCount - 1 ] = scaledResultBuffer[ idx ];

	activeTransmissions[ idx ] = true;

}

void __device__ finishTransmission( int idx,
		int* transmissionBins,
		hipfftReal* transmissionEnds,
		bool* activeTransmissions
	)
{

	for( int i = transmissionCount - 1 ; i >= 0 ; i-- )
	{

		if( transmissionBins[ i ] == idx )
		{

			transmissionEnds[ i ] = timeStep / SAMPLE_RATE;

			activeTransmissions[ idx ] = false;

			return;

		}

	}

}

void __global__ scaleResult( hipfftReal* scaledResultBuffer , hipfftComplex* resultBuffer )
{

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if( idx < BLOCK_SIZE )
	{

		scaledResultBuffer[ idx ] = sqrt( resultBuffer[ idx ].x * resultBuffer[ idx ].x * +
										  resultBuffer[ idx ].y * resultBuffer[ idx ].y      );

		scaledResultBuffer[ idx ] = 20 * log10( scaledResultBuffer[ idx ] );

	}

}

void __global__ initTransmissionArray( bool* activeTransmissions )
{

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	transmissionCount = 0;

	if( idx < BLOCK_SIZE )

		activeTransmissions[ idx ] = false;

}

void __global__ findTransmissions(
		hipfftReal* scaledResultBuffer ,
		int* deviceBins,
		hipfftReal *deviceFrequencies,
		hipfftReal *deviceStarts,
		hipfftReal *deviceEnds,
		hipfftReal *deviceStrengths,
		bool* activeTransmissions
	)
{

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if( idx < BLOCK_SIZE && idx != 0 )
	{

		if( scaledResultBuffer[ idx ] > SIGNAL_THRESHOLD && activeTransmissions[ idx ] == false )
		{

			atomicAdd( &transmissionCount , 1 );

			createTransmission( idx , deviceBins , scaledResultBuffer , deviceFrequencies , deviceStarts , deviceStrengths, activeTransmissions );

		}

		if( scaledResultBuffer[ idx ] < SIGNAL_THRESHOLD && activeTransmissions[ idx ] == true )
		{

			finishTransmission( idx , deviceBins , deviceEnds , activeTransmissions );

		}

	}

	//timeStep += BLOCK_SIZE;

	atomicAdd( &timeStep , 1 );

}

///////////////////////////////////////////////////////////////////////////////////////////////
////
////    Device Pointers
////
///////////////////////////////////////////////////////////////////////////////////////////////

hipfftComplex *deviceResult = 0;

hipfftReal *deviceSource = 0;

hipfftReal *deviceScaledResult = 0;

int* deviceBins = 0;

hipfftReal *deviceFrequencies = 0;

hipfftReal *deviceStarts = 0;

hipfftReal *deviceEnds = 0;

hipfftReal *deviceStrengths = 0;

bool* deviceActiveTransmissions = 0;

int* deviceCount = 0;

///////////////////////////////////////////////////////////////////////////////////////////////
////
////    Host Variables
////
///////////////////////////////////////////////////////////////////////////////////////////////

int* hostBins;

hipfftReal *hostFrequencies;

hipfftReal *hostStarts;

hipfftReal *hostEnds;

hipfftReal *hostStrengths;

///////////////////////////////////////////////////////////////////////////////////////////////
////
////    Functions
////
///////////////////////////////////////////////////////////////////////////////////////////////

int main( int argc , char** argv )
{
	
	std::string filename = std::string( argv[1] );

	std::ifstream f;

	struct stat filestatus;

	stat( filename.c_str() , &filestatus );

	size_t filesize = filestatus.st_size;

	f.open( filename.c_str() , std::ios::in | std::ios::binary );

	if( !f.good() )
	{
	
		std::cerr << "Can't open file" << std::endl;

		exit( 1 );

	}

	hipfftReal* original = new hipfftReal[ filesize ];

	for( unsigned int i = 0 ; i < filesize ; i++ )
	{
	
		original[i] = (hipfftReal) (byte) f.get();

	}

	f.close();

	int fft_size = BLOCK_SIZE;

	int max_transmissions = MAX_TRANSMISSIONS;

	hipGetSymbolAddress( (void**) &deviceCount , HIP_SYMBOL(transmissionCount) );

	gpuErrchk( hipMalloc( &deviceSource , filesize * sizeof(hipfftReal) ));

	gpuErrchk( hipMalloc( &deviceResult ,  fft_size * sizeof(hipfftComplex) ));

	gpuErrchk( hipMalloc( &deviceScaledResult ,  fft_size * sizeof(hipfftReal) ));

	gpuErrchk( hipMalloc( &deviceBins , max_transmissions * sizeof(int) ));

	gpuErrchk( hipMalloc( &deviceFrequencies ,  max_transmissions * sizeof(hipfftReal) ));

	gpuErrchk( hipMalloc( &deviceStarts , max_transmissions * sizeof(hipfftReal) ));

	gpuErrchk( hipMalloc( &deviceEnds ,  max_transmissions * sizeof(hipfftReal) ));

	gpuErrchk( hipMalloc( &deviceStrengths ,  max_transmissions * sizeof(hipfftReal) ));

	gpuErrchk( hipMalloc( &deviceActiveTransmissions ,  fft_size * sizeof(bool) ));

	// TODO: This giant memcpy will become a pipelined streaming thingy
	gpuErrchk( hipMemcpy( deviceSource , original , filesize * sizeof( hipfftReal ) , hipMemcpyHostToDevice ));

	initTransmissionArray<<< 64 , 32 >>>( deviceActiveTransmissions );

	//prepare the FFT
	hipfftHandle p;

	hipfftResult_t fft_result;

	fft_result = hipfftPlan1d( &p , BLOCK_SIZE , HIPFFT_R2C , BATCH_SIZE );

	if( fft_result != HIPFFT_SUCCESS )

		exit(1);


	for( unsigned int j = 0 ; j < filesize * 0.25 - fft_size  ; j += fft_size )
	{
		
		fft_result = hipfftExecR2C( p , deviceSource + j * sizeof( hipfftReal ) , deviceResult );
		
		if( fft_result != HIPFFT_SUCCESS )

			exit(2);

		// num blocks * num threads = fftsize / 2 ... nyquist limit
		scaleResult<<< 64 , 32 >>>( deviceScaledResult , deviceResult );

		gpuErrchk( hipPeekAtLastError() );

		findTransmissions<<< 64 , 32 >>>(
				deviceScaledResult,
				deviceBins,
				deviceFrequencies,
				deviceStarts,
				deviceEnds,
				deviceStrengths,
				deviceActiveTransmissions
			);

		gpuErrchk( hipPeekAtLastError() );

	}

	//Copy all that crap back
	int* hostCount = new int;

	hostBins = new int[ MAX_TRANSMISSIONS ];

	hostFrequencies = new hipfftReal[ MAX_TRANSMISSIONS ];

	hostStarts = new hipfftReal[ MAX_TRANSMISSIONS ];

	hostEnds = new hipfftReal[ MAX_TRANSMISSIONS ];

	hostStrengths = new hipfftReal[ MAX_TRANSMISSIONS ];

	std::cout << "LOLZ" << std::endl;

	gpuErrchk( hipMemcpy( hostBins , deviceBins , MAX_TRANSMISSIONS * sizeof( int ) , hipMemcpyDeviceToHost ));

	gpuErrchk( hipMemcpy( hostFrequencies , deviceFrequencies , MAX_TRANSMISSIONS * sizeof( hipfftReal ) , hipMemcpyDeviceToHost ));

	gpuErrchk( hipMemcpy( hostStarts , deviceStarts , MAX_TRANSMISSIONS * sizeof( hipfftReal ) , hipMemcpyDeviceToHost ));

	gpuErrchk( hipMemcpy( hostEnds , deviceEnds , MAX_TRANSMISSIONS * sizeof( hipfftReal ) , hipMemcpyDeviceToHost ));

	gpuErrchk( hipMemcpy( hostStrengths , deviceStrengths , MAX_TRANSMISSIONS * sizeof( hipfftReal ) , hipMemcpyDeviceToHost ));

	gpuErrchk( hipMemcpy( hostCount , deviceCount , sizeof( int ) , hipMemcpyDeviceToHost ) );

	std::cout << *hostCount << std::endl;

	std::ofstream fo;

	fo.open( "spikes.txt" );

	for( unsigned int i = 0 ; i < *hostCount ; i++ )
	{

		fo << "==== TRANSMISSION ====" << "\n";

		//In MHz
		fo << "Bin             : " << hostBins[ i ] << " \n";
		fo << "Frequency       : " << hostFrequencies[ i ] << " MHz\n";
		fo << "Signal strength : " << hostStrengths[ i ] << " dB\n";
		fo << "Time start      : " << hostStarts[ i ] << " s\n";
		fo << "Time end        : " << hostEnds[ i ] << " s\n";

	}

	return 0;

}
